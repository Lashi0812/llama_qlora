#include "hip/hip_runtime.h"
#include "include/quantize.cuh"
#include <iostream>
#include <vector>
#include <ATen/ATen.h>
#include <torch/extension.h>

// std::vector<int> Dummy(int const N) {
//     return std::vector<int>(N,N);
// }

// std::vector<at::Tensor> Dummy2(int const N) {
//     return std::vector<at::Tensor>{at::ones({1,2}),at::zeros({2,2})};
// }

std::vector<at::Tensor> quantizeNF4(const at::Tensor weights, const int blockSize)
{
    // get number of element
    auto nElems = weights.numel();

    // find the execution configuration
    unsigned int gridSize = (nElems + blockSize - 1) / blockSize;
    

    // create out and absmax tensor which we need to pass into the kernel
    at::Tensor absmax = at::zeros({
                                      gridSize,
                                  },
                                  at::TensorOptions().device(at::kCUDA, 0).dtype(at::kFloat));
    at::Tensor out = at::zeros({(nElems + 1) / 2, 1}, at::TensorOptions().device(at::kCUDA, 0).dtype(at::kByte));

    // launch the kernel
    kQuantizeNF4<__hip_bfloat16,2,64><<<gridSize, 32>>>(reinterpret_cast<__hip_bfloat16*>(weights.data_ptr()), absmax.data_ptr<float>(), out.data_ptr<unsigned char>(), nElems);
    return std::vector<at::Tensor>{absmax, out};
}

at::Tensor deQuantizeNF4(const at::Tensor quant,const at::Tensor absmax,const at::IntArrayRef outShape)
{
    at::Tensor out = at::zeros(outShape,at::TensorOptions().device(at::kCUDA,0).dtype(at::kBFloat16));
    // launch execution configuration 
    int blockSize = (quant.numel() / absmax.numel())/2;
    dim3 block(blockSize);
    dim3 grid(absmax.numel(),1);

    // launch the kernel
    kDequantizeNF4<__hip_bfloat16,2,32><<<grid,16>>>(absmax.data_ptr<float>(),quant.data_ptr<unsigned char>(),reinterpret_cast<__hip_bfloat16*>(out.data_ptr()));

    return out;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m)
{
    // m.def("dummy",&Dummy,"pybind");
    // m.def("dummy2",&Dummy2,"pybind");
    // m.def("cuda_hello",&cuda_hello,"cuda bind");
    m.def("quantizeNF4", &quantizeNF4, "Quantize bf16 to NF4 format");
    m.def("dequantizeNF4", &deQuantizeNF4, "Dequantize to bf16");
}
int main()
{
    auto a = at::rand({4096,4096},at::TensorOptions().dtype(at::kBFloat16).device(at::kCUDA,0));
    auto ans = quantizeNF4(a,64);
    std::cout << ans[0] << std::endl;
    std::cout << ans[1] << std::endl;
}

